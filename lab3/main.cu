#include "hip/hip_runtime.h"
﻿#include "json.hpp"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>

using namespace std;
using json = nlohmann::json;


#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, const char* func, const char* file, int line) {
    if (result) {
        std::cout << "CUDA error: " << hipGetErrorString(result) << " (error code " << static_cast<unsigned int>(result) << ")";
        std::cout << " at " << file << ":" << line << " '" << func << "' \n";
        hipDeviceReset();
        exit(-1);
    }
}

struct DataEntry {
    char name[64];
    float sugar;
    int criteria;
};

struct Result {
    char text[64];
};

__device__ static size_t get_string_size(char *text) {
    size_t size = 0;
    while (text[size] != 0) {
        size++;
    }
    return size;
}

__global__ void processEntriesKernel(DataEntry *entries, Result *results, size_t entry_count)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= entry_count) return;

    auto entry = &entries[idx];
    if (entry->sugar < entry->criteria) return;
    auto result = &results[idx];
 
    for (int i = 0; i < get_string_size(entry->name)/2; i++) {
        result->text[i] = entry->name[2*i];
        if ('a' <= result->text[i] && result->text[i] <= 'z') {
            result->text[i] -= 32; // Convert to uppercase
        }
    }
}

int main(int argc, char** argv)
{
    int block_count = 8;
    int block_size  = 32;
    const char* input_path = "IF-1-1_PuzonasR_L3_dat_1.json";
    const char* output_path = "output.txt";

    std::ifstream f(input_path);
    json data = json::parse(f);

    vector<DataEntry> entries;
    for (auto &it : data) {
        auto entry = DataEntry{ 0 };
        strcpy(entry.name, it["name"].get<std::string>().c_str());
        entry.sugar = it["sugar"];
        entry.criteria = it["criteria"];
        entries.push_back(entry);
    }

    int entry_count = entries.size();
    cout << "Input data count: " << entry_count << endl;

    if (entry_count > block_count * block_size) {
        cout << "WARNING! Not enough blocks/threads, the total number threads is " << block_count * block_size << ", but you need " << entry_count << endl;
    }

    DataEntry* device_entries = NULL;
    Result* device_results = NULL;
    checkCudaErrors(hipMalloc((void**)&device_entries, entry_count * sizeof(DataEntry)));
    checkCudaErrors(hipMalloc((void**)&device_results, entry_count * sizeof(Result)));

    checkCudaErrors(hipMemcpy(device_entries, &entries[0], entry_count * sizeof(DataEntry), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(device_results, 0, entry_count * sizeof(Result)));
    checkCudaErrors(hipDeviceSynchronize());

    processEntriesKernel<<<block_count, block_size>>>(device_entries, device_results, entry_count);
    checkCudaErrors(hipDeviceSynchronize());

    Result* results = (Result*)malloc(entry_count * sizeof(Result));
    checkCudaErrors(hipMemcpy(results, device_results, entry_count * sizeof(Result), hipMemcpyDeviceToHost));

    int result_count = 0;
    std::ofstream output_file(output_path);
    for (int i = 0; i < entry_count; i++) {
        if (results[i].text[0] == 0) continue;

        output_file << results[i].text << endl;
        result_count++;
    }

    cout << "Result data count: " << result_count << endl;

    free(results);
    hipFree(device_entries);
    hipFree(device_results);
    return 0;
}
